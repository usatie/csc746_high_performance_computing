#include "hip/hip_runtime.h"
//
// (C) 2021, E. Wes Bethel
// sobel_gpu.cpp
// usage:
//      sobel_gpu [no args, all is hard coded]
//

#include <iostream>
#include <iomanip>
#include <vector>
#include <chrono>
#include <unistd.h>
#include <string.h>
#include <math.h>

// see https://en.wikipedia.org/wiki/Sobel_operator


// easy-to-find and change variables for the input.
// specify the name of a file containing data to be read in as bytes, along with 
// dimensions [columns, rows]

// this is the original laughing zebra image
//static char input_fname[] = "../data/zebra-gray-int8";
//static int data_dims[2] = {3556, 2573}; // width=ncols, height=nrows
//char output_fname[] = "../data/processed-raw-int8-cpu.dat";

// this one is a 4x augmentation of the laughing zebra
static char input_fname[] = "../data/zebra-gray-int8-4x";
static int data_dims[2] = {7112, 5146}; // width=ncols, height=nrows
char output_fname[] = "../data/processed-raw-int8-4x-gpu.dat";

// see https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
// macro to check for cuda errors. basic idea: wrap this macro around every cuda call
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//
// this function is callable only from device code
//
// perform the sobel filtering at a given i,j location
// input: float *s - the source data
// input: int i,j - the location of the pixel in the source data where we want to center our sobel convolution
// input: int nrows, ncols: the dimensions of the input and output image buffers
// input: float *gx, gy:  arrays of length 9 each, these are logically 3x3 arrays of sobel filter weights
//
// this routine computes Gx=gx*s centered at (i,j), Gy=gy*s centered at (i,j),
// and returns G = sqrt(Gx^2 + Gy^2)

// see https://en.wikipedia.org/wiki/Sobel_operator
//
__device__ float
sobel_filtered_pixel(const float *s, int i, int j , int ncols, int nrows, const float *gx, const float *gy)
{

   float t=0.0;

   // ADD CODE HERE:  add your code here for computing the sobel stencil computation at location (i,j)
   // of input s, returning a float
   float Gx = 0.0, Gy = 0.0;
   for (int ii = 0; ii < 3; ++ii) {
           for (int jj = 0; jj < 3; ++jj) {
        	   int r = i - 1 + ii;
        	   int c = j - 1 + jj;
        	   if (r < 0 || r >= nrows || c < 0 || c >= ncols)
        		   return 0;
		   if (r * ncols + c >= ncols * nrows)
			   return 0;
		   int s_index = r * ncols + c;
		   int g_index = ii * 3 + jj;
        	   Gx += s[s_index] * gx[g_index];
        	   Gy += s[s_index] * gy[g_index];
           }
   }
   t = sqrt(Gx * Gx + Gy * Gy);

   return t;
}

//
// this function is the kernel that runs on the device
// 
// this code will look at CUDA variables: blockIdx, blockDim, threadIdx, blockDim and gridDim
// to compute the index/stride to use in striding through the source array, calling the
// sobel_filtered_pixel() function at each location to do the work.
//
// input: float *s - the source data, size=rows*cols
// input: int i,j - the location of the pixel in the source data where we want to center our sobel convolution
// input: int nrows, ncols: the dimensions of the input and output image buffers
// input: float *gx, gy:  arrays of length 9 each, these are logically 3x3 arrays of sobel filter weights
// output: float *d - the buffer for the output, size=rows*cols.
//


__global__ void
sobel_kernel_gpu(const float *s,  // source image pixels
      float *d,  // dst image pixels
      int n,  // size of image cols*rows,
      int nrows,
      int ncols,
      const float *gx, const float *gy) // gx and gy are stencil weights for the sobel filter
{
   // ADD CODE HERE: insert your code here that iterates over every (i,j) of input,  makes a call
   // to sobel_filtered_pixel, and assigns the resulting value at location (i,j) in the output.
   // because this is CUDA, you need to use CUDA built-in variables to compute an index and stride
   // your processing motif will be very similar here to that we used for vector add in Lab #2
   int width, height;

   width=ncols;
   height=nrows;
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   int stride = blockDim.x * gridDim.x;
   for (int i = index; i < height; i += stride)
       for (int j = 0; j < width; j++)
          d[i * width + j] = sobel_filtered_pixel(s, i, j, width, height, gx, gy);
}

int
main (int ac, char *av[])
{
   // input, output file names hard coded at top of file

   // load the input file
   off_t nvalues = data_dims[0]*data_dims[1];
   unsigned char *in_data_bytes = (unsigned char *)malloc(sizeof(unsigned char)*nvalues);

   FILE *f = fopen(input_fname,"r");
   if (fread((void *)in_data_bytes, sizeof(unsigned char), nvalues, f) != nvalues*sizeof(unsigned char))
   {
      printf("Error reading input file. \n");
      fclose(f);
      return 1;
   }
   else
      printf(" Read data from the file %s \n", input_fname);
   fclose(f);

#define ONE_OVER_255 0.003921568627451

   // now convert input from byte, in range 0..255, to float, in range 0..1
   float *in_data_floats;
   gpuErrchk( hipMallocManaged(&in_data_floats, sizeof(float)*nvalues) );

   for (off_t i=0; i<nvalues; i++)
      in_data_floats[i] = (float)in_data_bytes[i] * ONE_OVER_255;

   // now, create a buffer for output
   float *out_data_floats;
   gpuErrchk( hipMallocManaged(&out_data_floats, sizeof(float)*nvalues) );
   for (int i=0;i<nvalues;i++)
      out_data_floats[i] = 1.0;  // assign "white" to all output values for debug

   // define sobel filter weights, copy to a device accessible buffer
   float Gx[9] = {1.0, 0.0, -1.0, 2.0, 0.0, -2.0, 1.0, 0.0, -1.0};
   float Gy[9] = {1.0, 2.0, 1.0, 0.0, 0.0, 0.0, -1.0, -2.0, -1.0};
   float *device_gx, *device_gy;
   gpuErrchk( hipMallocManaged(&device_gx, sizeof(Gx)) );
   gpuErrchk( hipMallocManaged(&device_gy, sizeof(Gy)) );

   for (int i=0;i<9;i++) // copy from Gx/Gy to device_gx/device_gy
   {
      device_gx[i] = Gx[i];
      device_gy[i] = Gy[i];
   }
   
   // now, induce memory movement to the GPU of the data in unified memory buffers

   int deviceID=0; // assume GPU#0, always. OK assumption for this program
   hipMemPrefetchAsync((void *)in_data_floats, nvalues*sizeof(float), deviceID);
   hipMemPrefetchAsync((void *)out_data_floats, nvalues*sizeof(float), deviceID);
   hipMemPrefetchAsync((void *)device_gx, sizeof(Gx), deviceID);
   hipMemPrefetchAsync((void *)device_gy, sizeof(Gy), deviceID);

   // set up to run the kernel
   int nBlocks=1, nThreadsPerBlock=256;

   // ADD CODE HERE: insert your code here to set a different number of thread blocks or # of threads per block
   // nBlocks = (nvalues + nThreadsPerBlock - 1) / nThreadsPerBlock;


   printf(" GPU configuration: %d blocks, %d threads per block \n", nBlocks, nThreadsPerBlock);

   // start timer
   std::cout << std::fixed << std::setprecision(6);
   std::chrono::time_point<std::chrono::high_resolution_clock> start_time = std::chrono::high_resolution_clock::now();
   // invoke the kernel on the device
   sobel_kernel_gpu<<<nBlocks, nThreadsPerBlock>>>(in_data_floats, out_data_floats, nvalues, data_dims[1], data_dims[0], device_gx, device_gy);
   // wait for it to finish, check errors
   gpuErrchk (  hipDeviceSynchronize() );

   // end timer
   std::chrono::time_point<std::chrono::high_resolution_clock> end_time = std::chrono::high_resolution_clock::now();
   std::chrono::duration<double> elapsed = end_time - start_time;
   std::cout << " Elapsed time is : " << elapsed.count() << " (sec) " << std::endl;

   // write output after converting from floats in range 0..1 to bytes in range 0..255
   unsigned char *out_data_bytes = in_data_bytes;  // just reuse the buffer from before
   for (off_t i=0; i<nvalues; i++)
      out_data_bytes[i] = (unsigned char)(out_data_floats[i] * 255.0);

   f = fopen(output_fname,"w");

   if (fwrite((void *)out_data_bytes, sizeof(unsigned char), nvalues, f) != nvalues*sizeof(unsigned char))
   {
      printf("Error writing output file. \n");
      fclose(f);
      return 1;
   }
   else
      printf(" Wrote the output file %s \n", output_fname);
   fclose(f);
}

// eof

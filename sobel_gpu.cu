#include "hip/hip_runtime.h"
//
// (C) 2021, E. Wes Bethel
// sobel_gpu.cpp
// usage:
//      sobel_gpu [no args, all is hard coded]
//

#include <chrono>
#include <iomanip>
#include <iostream>
#include <math.h>
#include <string.h>
#include <unistd.h>
#include <vector>

// see https://en.wikipedia.org/wiki/Sobel_operator

// easy-to-find and change variables for the input.
// specify the name of a file containing data to be read in as bytes, along with
// dimensions [columns, rows]

// this is the original laughing zebra image
// static char input_fname[] = "../data/zebra-gray-int8";
// static int data_dims[2] = {3556, 2573}; // width=ncols, height=nrows
// char output_fname[] = "../data/processed-raw-int8-cpu.dat";

// this one is a 4x augmentation of the laughing zebra
static char input_fname[] = "../data/zebra-gray-int8-4x";
static int data_dims[2] = {7112, 5146}; // width=ncols, height=nrows
char output_fname[] = "../data/processed-raw-int8-4x-gpu.dat";

// see
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
// macro to check for cuda errors. basic idea: wrap this macro around every cuda
// call
#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

__constant__ float device_gx[9];
__constant__ float device_gy[9];

// this function is callable only from device code
//
// sobel_filtered_pixel(): perform the sobel filtering at a given x,y location
//
// input: float *s - the source data
// input: int x,y - the location of the pixel in the source data where we want to center our sobel convolution
// input: int nrows, ncols: the dimensions of the input and output image buffers
// input: float *gx, gy:  arrays of length 9 each, these are logically 3x3 arrays of sobel filter weights
//
// this routine computes Gx=gx*s centered at (x,y), Gy=gy*s centered at (x,y),
// and returns G = sqrt(Gx^2 + Gy^2)
__device__
float sobel_filtered_pixel(const float *s, int x, int y, int ncols, int nrows) {
  float t = 0.0;
  
  // if x or y is at the boundary of the img or out of the boundary, we can't compute
  if (x <= 0 || x >= ncols - 1 || y <= 0 || y >= nrows - 1)
	  return t;

  // ADD CODE HERE: add your code here for computing the sobel stencil
  // computation at location (x,y) of input s, returning a float
  float Gx = 0.0, Gy = 0.0;
  for (int i = 0; i < 3; ++i) {
    for (int j = 0; j < 3; ++j) {
      int xx = x - 1 + i;
      int yy = y - 1 + j;
      // Gx += s[xx, yy] * gx[i, j]
      Gx += s[yy * ncols + xx] * device_gx[j * 3 + i];
      // Gy += s[xx, yy] * gy[i, j]
      Gy += s[yy * ncols + xx] * device_gy[j * 3 + i];
    }
  }
  t = sqrt(Gx * Gx + Gy * Gy);
  return t;
}

//
// this function is the kernel that runs on the device
//
// this code will look at CUDA variables: blockIdx, blockDim, threadIdx,
// blockDim and gridDim to compute the index/stride to use in striding through
// the source array, calling the sobel_filtered_pixel() function at each
// location to do the work.
//
// input: float *s - the source data, size=rows*cols
// input: int i,j - the location of the pixel in the source data where we want
// to center our sobel convolution input: int nrows, ncols: the dimensions of
// the input and output image buffers input: float *gx, gy:  arrays of length 9
// each, these are logically 3x3 arrays of sobel filter weights output: float *d
// - the buffer for the output, size=rows*cols.
//

__global__ void sobel_kernel_gpu(
    const float *s, // source image pixels
    float *d,       // dst image pixels
    int n,          // size of image cols*rows,
    int ncols, int nrows)
{
  // ADD CODE HERE: insert your code here that iterates over every (i,j) of
  // input,  makes a call to sobel_filtered_pixel, and assigns the resulting
  // value at location (i,j) in the output. because this is CUDA, you need to
  // use CUDA built-in variables to compute an index and stride your processing
  // motif will be very similar here to that we used for vector add in Lab #2
  int width, height;

  width = ncols;
  height = nrows;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int y = index; y < height; y += stride)
    for (int x = 0; x < width; x++)
      d[y * width + x] = sobel_filtered_pixel(s, x, y, width, height);
}

int main(int ac, char *av[]) {
  // input, output file names hard coded at top of file

  // load the input file
  off_t nvalues = data_dims[0] * data_dims[1];
  unsigned char *in_data_bytes =
      (unsigned char *)malloc(sizeof(unsigned char) * nvalues);

  FILE *f = fopen(input_fname, "r");
  if (fread((void *)in_data_bytes, sizeof(unsigned char), nvalues, f) !=
      nvalues * sizeof(unsigned char)) {
    printf("Error reading input file. \n");
    fclose(f);
    return 1;
  } else
    printf(" Read data from the file %s \n", input_fname);
  fclose(f);

#define ONE_OVER_255 0.003921568627451

  // now convert input from byte, in range 0..255, to float, in range 0..1
  float *in_data_floats;
  gpuErrchk(hipMallocManaged(&in_data_floats, sizeof(float) * nvalues));

  for (off_t i = 0; i < nvalues; i++)
    in_data_floats[i] = (float)in_data_bytes[i] * ONE_OVER_255;

  // now, create a buffer for output
  float *out_data_floats;
  gpuErrchk(hipMallocManaged(&out_data_floats, sizeof(float) * nvalues));
  for (int i = 0; i < nvalues; i++)
    out_data_floats[i] = 1.0; // assign "white" to all output values for debug

  // define sobel filter weights, copy to a device accessible buffer
  float Gx[9] = {1.0, 0.0, -1.0, 2.0, 0.0, -2.0, 1.0, 0.0, -1.0};
  float Gy[9] = {1.0, 2.0, 1.0, 0.0, 0.0, 0.0, -1.0, -2.0, -1.0};
  gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(device_gx), Gx, sizeof(Gx)) );
  gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(device_gy), Gy, sizeof(Gy)) );

  // now, induce memory movement to the GPU of the data in unified memory
  // buffers

  int deviceID = 0; // assume GPU#0, always. OK assumption for this program
  hipMemPrefetchAsync((void *)in_data_floats, nvalues * sizeof(float),
                       deviceID);
  hipMemPrefetchAsync((void *)out_data_floats, nvalues * sizeof(float),
                       deviceID);

  // set up to run the kernel
  int nBlocks = 1, nThreadsPerBlock = 256;

  // ADD CODE HERE: insert your code here to set a different number of thread
  // blocks or # of threads per block
  int c;
  while ( (c = getopt(ac, av, "T:B:")) != -1) {
     switch(c) {
        case 'T':
           nThreadsPerBlock = std::atoi(optarg == NULL ? "-999" : optarg);
           std::cout << "Command line number of `threads per block`: " << nThreadsPerBlock << std::endl;
           break;
        case 'B':
           nBlocks = std::atoi(optarg == NULL ? "-999" : optarg);
           std::cout << "Command line number of `thread blocks`: " << nBlocks << std::endl;
           break;
     }
  }
  

  printf(" GPU configuration: %d blocks, %d threads per block \n", nBlocks,
         nThreadsPerBlock);

  // start timer
  std::cout << std::fixed << std::setprecision(6);
  std::chrono::time_point<std::chrono::high_resolution_clock> start_time =
      std::chrono::high_resolution_clock::now();
  // invoke the kernel on the device
  sobel_kernel_gpu<<<nBlocks, nThreadsPerBlock>>>(
      in_data_floats, out_data_floats, nvalues, data_dims[0], data_dims[1]);
  // wait for it to finish, check errors
  gpuErrchk(hipDeviceSynchronize());

  // end timer
  std::chrono::time_point<std::chrono::high_resolution_clock> end_time =
      std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = end_time - start_time;
  std::cout << " Elapsed time is : " << elapsed.count() << " (sec) "
            << std::endl;

  // write output after converting from floats in range 0..1 to bytes in range
  // 0..255
  unsigned char *out_data_bytes =
      in_data_bytes; // just reuse the buffer from before
  for (off_t i = 0; i < nvalues; i++)
    out_data_bytes[i] = (unsigned char)(out_data_floats[i] * 255.0);

  f = fopen(output_fname, "w");

  if (fwrite((void *)out_data_bytes, sizeof(unsigned char), nvalues, f) !=
      nvalues * sizeof(unsigned char)) {
    printf("Error writing output file. \n");
    fclose(f);
    return 1;
  } else
    printf(" Wrote the output file %s \n", output_fname);
  fclose(f);
}

// eof
